#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <helper_timer.h>
#define NUM_BANKS 32
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif
int BLOCK_SIZE = 1024;
int SEGMENT_SIZE = 2 * BLOCK_SIZE;

//Define functions...
hipError_t fullScan(int* out, int* in, const int size); 
hipError_t fullScan2(int* out, int* in, const int size);

//Prescan with bank conflict avoidance...
__global__ void prescan2(int* outputData, int* inputData, int n, int ss) {
	extern __shared__ int temp[]; //Allocated on invocation - Pointer to shared memory

	//ThreadId - 0 --> total number of threads provided..
	int threadID = threadIdx.x;
	int gThreadID = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = 1;

	//Max data access - 2x thread id.
	int ai = gThreadID;
	int bi = gThreadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(ai);
	if (2 * gThreadID < n) {
		temp[ai + bankOffsetA] = inputData[ai];
		temp[bi + bankOffsetB] = inputData[bi];
	}

	for (int d = ss >> 1; d > 0; d >>= 1) { //Build sum in place up the tree
		__syncthreads();

		if (threadID < d) {
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}


	//Clear the last element
	if (threadID == 0) {
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}

	//Traverse the tree and build scan
	for (int d = 1; d < ss; d *= 2) {
		offset >>= 1;
		__syncthreads();

		if (threadID < d) {
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();

	//Write the results to the device memory
	if (2 * gThreadID < n) {
		outputData[ai] = temp[ai + bankOffsetA];
		outputData[bi] = temp[bi + bankOffsetB];
	}
}

//Std prescan...
//outputData - output array, inputData - input array, n- arraysize, ss- segment size
__global__ void prescan(int* outputData, int* inputData, int n, int ss) {
	extern __shared__ int temp[]; //Allocated on invocation - Pointer to shared memory

	//ThreadId - 0 --> total number of threads provided..
	int threadID = threadIdx.x;
	int offset = 1;
	int gThreadID = blockIdx.x * blockDim.x + threadIdx.x;

	//Max data access - 2x thread id.
	if (2 * gThreadID < n) {
		temp[2 * threadID] = inputData[2 * gThreadID]; //Load input into shared memory
		temp[2 * threadID + 1] = inputData[2 * gThreadID + 1];
	}

	for (int d = ss>> 1; d > 0; d >>= 1) { //Build sum in place up the tree
		__syncthreads();

		if (threadID < d) {
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}


	//Clear the last element
	if (threadID == 0) {
		temp[ss - 1] = 0;
	}

	//Traverse the tree and build scan
	for (int d = 1; d < ss; d *= 2) {
		offset >>= 1;
		__syncthreads();

		if (threadID < d) {
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();

	//Write the results to the device memory
	if (2* gThreadID < n) {
		outputData[2 * gThreadID] = temp[2 * threadID];
		outputData[2 * gThreadID + 1] = temp[2 * threadID + 1];
	}
}



//Main Method
int main()
{
	//Initialise array...k
	const int arraySize = 10000;

	int inputArray[arraySize];
	int inputArray2[arraySize];
	int outputArray[arraySize];
	int outputArray2[arraySize];

	//Create array to input...
	for (int i = 0; i < arraySize; i++) {
		inputArray[i] = 1;
		printf("%d ", inputArray[i]);
	}

	//Create array to input...
	for (int i = 0; i < arraySize; i++) {
		inputArray2[i] = 1;
		printf("%d ", inputArray2[i]);
	}

    // Add vectors in parallel.
    hipError_t cudaStatus = fullScan(outputArray, inputArray, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Prescan failed!");
        return 1;
    }

	//Output results
	printf("\n\n");
	for(int i = 0; i < arraySize; i++) {
		printf("%d ", outputArray[i]);
	}

	// Add vectors in parallel - Bank conflict Avoidance
	hipError_t cudaStatus2 = fullScan2(outputArray2, inputArray2, arraySize);
	if (cudaStatus2 != hipSuccess) {
		fprintf(stderr, "Prescan failed! (bank conflict avoid)");
		return 1;
	}

	//Output results
	printf("\n\n");
	for (int i = 0; i < arraySize; i++) {
		printf("%d ", outputArray2[i]);
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel. size - array size;
hipError_t fullScan(int *out, int* in, const int size)
{
	//Init Stopwatch
	StopWatchInterface* timer = NULL;
	double h_msecs = NULL;

	//Properties for prescan
	int inputVectorSize = size; //Size of the input vector (i.e. array)
	int threadsPerBlock = 1024;
	//int blocksPerGrid = 1 + (inputVectorSize - 1) / threadsPerBlock;
	int blocksPerGrid = ceil(inputVectorSize / (float)SEGMENT_SIZE);
	int sharedMemAmount = (SEGMENT_SIZE) * sizeof(int); //Amount of shared memory given to prescan 

	//Init cuda timer
	hipEvent_t start, stop;
	float d_msecs;

	//Initialise arrays
    int *dev_in = 0;
    int *dev_out = 0;
    
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .

    cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_in, in, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy input from system to GPU!");
        goto Error;
	}

	//Start Std Timer.
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	//Start Cuda Timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//Launch prescan.
	printf("\n Launching prescan - \nsize - %d seg size- %d", size, SEGMENT_SIZE);
	printf("\n BlocksPer Grid- %d, threadsPerBlock %d, sharedMemAmount %d", blocksPerGrid, threadsPerBlock, sharedMemAmount);
	prescan <<<blocksPerGrid,threadsPerBlock,sharedMemAmount>>> (dev_out, dev_in,size,SEGMENT_SIZE);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();

	//Stop timer.
	sdkStopTimer(&timer);
	h_msecs = sdkGetTimerValue(&timer); 
	printf("\n Standard Timer-  Ran in %.5f ms", h_msecs);
	sdkDeleteTimer(&timer);


	//Stop CUDA timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&d_msecs, start, stop);
	printf("\n CUDA Timer-  Ran in %.5f ms", d_msecs);

	hipEventDestroy(start);
	hipEventDestroy(stop);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy from GPU to Host!");
        goto Error;
    }

Error:
    hipFree(dev_in);
    hipFree(dev_out);
    
    return cudaStatus;
}

//FullScan2 - with Bank Conflict Avoidance Prescan...
hipError_t fullScan2(int* out, int* in, const int size)
{
	//Init Stopwatch
	StopWatchInterface* timer = NULL;
	double h_msecs = NULL;

	//Properties for prescan
	int inputVectorSize = size; //Size of the input vector (i.e. array)
	int threadsPerBlock = 1024;
	//int blocksPerGrid = 1 + (inputVectorSize - 1) / threadsPerBlock;
	int blocksPerGrid = ceil(inputVectorSize / (float)SEGMENT_SIZE);
	int sharedMemAmount = (SEGMENT_SIZE) * sizeof(int); //Amount of shared memory given to prescan 

	//Init cuda timer
	hipEvent_t start, stop;
	float d_msecs;

	//Initialise arrays
	int* dev_in = 0;
	int* dev_out = 0;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .

	cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, in, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Failed to copy input from system to GPU!");
		goto Error;
	}

	//Start Std Timer.
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	//Start Cuda Timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//Launch prescan.
	prescan2 << <blocksPerGrid, threadsPerBlock,sharedMemAmount>>> (dev_out, dev_in, size,SEGMENT_SIZE);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();

	//Stop timer.
	sdkStopTimer(&timer);
	h_msecs = sdkGetTimerValue(&timer);
	printf("\n Bank Conflict - Standard Timer-  Ran in %.5f ms", h_msecs);
	sdkDeleteTimer(&timer);


	//Stop CUDA timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&d_msecs, start, stop);
	printf("\n Bank Conflict - CUDA Timer-  Ran in %.5f ms", d_msecs);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(out, dev_out, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Failed to copy from GPU to Host!");
		goto Error;
	}

Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}

